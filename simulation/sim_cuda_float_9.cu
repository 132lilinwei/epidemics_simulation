#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <math.h>
#include <vector>

#include "CycleTimer.h"
#include "crun.h"
#include "rutil.h"


#define BLOCK_SIZE 16
#define HUB_BLOCK_SIZE 32

float toBW(int bytes, float sec) {
  return (float)(bytes) / (1024. * 1024. * 1024.) / sec;
}


int *hub_device;
bool *mask_device;
int *neighbor_device;
int *neighbor_start_device; 

float *initial_load_factor_device;
int *rat_count_device;
int *infectious_rat_count_device;
float* weight_result_device;

float *neighbor_accum_weight_result_device;
float *sum_weight_result_device;

extern "C" void init_cuda(state_t *s) {
    graph_t *g = s->g;
    int nnode = g->nnode;
    int nedge = g->nedge;
    int nhub = g->nhub;


    hipMalloc(&hub_device, sizeof(int) * nhub);
    hipMalloc(&mask_device, sizeof(bool) * nnode);
    hipMalloc(&neighbor_device, sizeof(int) * (nnode+nedge));
    hipMalloc(&neighbor_start_device, sizeof(int) * (nnode+1));

    hipMalloc(&initial_load_factor_device, sizeof(float) * nnode);
    hipMalloc(&rat_count_device, sizeof(int) * nnode);
    hipMalloc(&infectious_rat_count_device, sizeof(int) * nnode);
    hipMalloc(&weight_result_device, sizeof(float) * nnode);

    hipMalloc(&neighbor_accum_weight_result_device, sizeof(float) * (nnode+nedge));
    hipMalloc(&sum_weight_result_device, sizeof(float) * nnode);


    float *temp = (float *)malloc(sizeof(float) * (nnode));
    for (int i = 0; i < nnode; i++) {
        temp[i] = static_cast<float>(s->initial_load_factor[i]);
    }
    hipMemcpy(initial_load_factor_device, temp, sizeof(float) * nnode, hipMemcpyHostToDevice); 
    free(temp);
    
    hipMemcpy(hub_device, g->hub, sizeof(int) * nhub, hipMemcpyHostToDevice); 
    hipMemcpy(mask_device, g->mask, sizeof(bool) * nnode, hipMemcpyHostToDevice);
    hipMemcpy(neighbor_device, g->neighbor, sizeof(int) * (nnode+nedge), hipMemcpyHostToDevice);
    hipMemcpy(neighbor_start_device, g->neighbor_start,  sizeof(int) * (nnode+1), hipMemcpyHostToDevice); 

}

extern "C" void clean_cuda() {
    hipFree(hub_device);
    hipFree(mask_device);
    hipFree(neighbor_device);
    hipFree(neighbor_start_device);

    hipFree(initial_load_factor_device);
    hipFree(rat_count_device);
    hipFree(infectious_rat_count_device);
    hipFree(weight_result_device);

    hipFree(neighbor_accum_weight_result_device);
    hipFree(sum_weight_result_device);
}





__device__ __inline__ float mweight_kernel(float val, float optval) {
    float arg = 1.0 + COEFF * (val - optval);
    float lg = log(arg) * M_LOG2E;
    float denom = 1.0 + lg * lg;
    return 1.0/denom;
}



__device__ __inline__ float imbalance_density_kernel(float ldensity, float rdensity) {
    return (rdensity - ldensity) / (rdensity + ldensity);
}


__device__ __inline__ float neighbor_ilf_fast_kernel(float load_factor, float *initial_load_factor, int* share_rat_count, int* share_infectious_rat_count, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id

    int share_block_size = BLOCK_SIZE+2;
    int in_block_x = threadIdx.x + 1;
    int in_block_y = threadIdx.y + 1;
    int in_block_id = in_block_x + in_block_y * share_block_size;

    float sum = 0.0;
    float ldensity, rdensity;
    int remote_x, remote_y, remote_in_block_x, remote_in_block_y, remote_nid, remote_in_block_id;

    int outdegree = 4;
    if (x == 0) {
        outdegree--;
    }
    if (y == 0) {
        outdegree--;
    }
    if (x == width -1) {
        outdegree--;
    }
    if (y == height-1) {
        outdegree--;
    }

    ldensity = (share_rat_count[in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[in_block_id] / share_rat_count[in_block_id];
    //up
    remote_x = x;
    remote_y = y+1;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x;
    remote_in_block_y = in_block_y+1;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * share_block_size;

    if (remote_y < height) {
        rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        float r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }


   
    //down
    remote_x = x;
    remote_y = y-1;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x;
    remote_in_block_y = in_block_y-1;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * share_block_size;

    if (remote_y >= 0) {
        rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        float r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }

    //left
    remote_x = x-1;
    remote_y = y;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x-1;
    remote_in_block_y = in_block_y;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * share_block_size;

    if (remote_x >= 0) {
        rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        float r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }

    //right
    remote_x = x+1;
    remote_y = y;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x+1;
    remote_in_block_y = in_block_y;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * share_block_size;

    if (remote_x <width) {
        rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        float r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }

    // change to a new ilf, where each node has different initial base ilf
    float ilf = BASE_ILF * (initial_load_factor[nid] / load_factor) + ILF_VARIABILITY * (sum / outdegree);
    return ilf;
}


__device__ __inline__ float neighbor_ilf_hub_kernel(float load_factor, float *initial_load_factor, int *rat_count, int *infectious_rat_count, int *neighbor, int *neighbor_start, int nid, int max_outdegree) {
    int outdegree = neighbor_start[nid+1] - neighbor_start[nid] - 1;
    outdegree = min(outdegree, max_outdegree);
    int *start = &neighbor[neighbor_start[nid]+1];
    int i;
    float sum = 0.0;
    for (i = 0; i < outdegree; i++) {
       
        float ldensity = (rat_count[nid] == 0) ? 0.0 : 1.0 * infectious_rat_count[nid] / rat_count[nid];
        float rdensity = (rat_count[start[i]] == 0) ? 0.0 : 1.0 * infectious_rat_count[start[i]] / rat_count[start[i]];
        float r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 : imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }
    // change to a new ilf, where each node has different initial base ilf
    float ilf = BASE_ILF * (initial_load_factor[nid] / load_factor) + ILF_VARIABILITY * (sum / outdegree);
    return ilf;
}



__global__ void
compute_weight_hub_kernel(float load_factor, float *initial_load_factor, int* hub, int nhub, int *rat_count, int *infectious_rat_count, int *neighbor, int *neighbor_start, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nodeid = hub[x];
        float ilf = neighbor_ilf_hub_kernel(load_factor, initial_load_factor, rat_count, infectious_rat_count, neighbor, neighbor_start, nodeid, INT_MAX); // INT_MAX means compute for all possible neighbors
        int count = rat_count[nodeid];
        result[nodeid] = mweight_kernel((float) count/load_factor, ilf);
    }
}



__global__ void
compute_weight_kernel(bool *mask, float load_factor, float *initial_load_factor, int *rat_count, int *infectious_rat_count, int *neighbor, int *neighbor_start, float* result, int width, int height) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thread_index = y * width + x; // thread_index is node id

    int share_block_size = BLOCK_SIZE+2;
    int in_block_x = threadIdx.x+1;
    int in_block_y = threadIdx.y+1;
    int in_block_id = in_block_x + in_block_y * share_block_size;


    __shared__ int share_rat_count[(BLOCK_SIZE+2) * (BLOCK_SIZE+2)];
    __shared__ int share_infectious_rat_count[(BLOCK_SIZE+2) * (BLOCK_SIZE+2)];

    share_rat_count[in_block_id] = rat_count[thread_index];
    share_infectious_rat_count[in_block_id] = infectious_rat_count[thread_index];

    if (threadIdx.x == 0 && x > 0) {
        share_rat_count[in_block_id-1] = rat_count[thread_index-1];
        share_infectious_rat_count[in_block_id-1] = infectious_rat_count[thread_index-1];
    }
    if (threadIdx.x == BLOCK_SIZE-1 && x < width - 1) {
        share_rat_count[in_block_id+1] = rat_count[thread_index+1];
        share_infectious_rat_count[in_block_id+1] = infectious_rat_count[thread_index+1];
    }
    if (threadIdx.y == 0 && y > 0) {
        share_rat_count[in_block_id - share_block_size] = rat_count[thread_index - width];
        share_infectious_rat_count[in_block_id - share_block_size] = infectious_rat_count[thread_index - width];
    }
    if (threadIdx.y == BLOCK_SIZE-1 && y < height - 1) {
        share_rat_count[in_block_id + share_block_size] = rat_count[thread_index + width];
        share_infectious_rat_count[in_block_id + share_block_size] = infectious_rat_count[thread_index + width];
    }



    if (x < width && y < height && mask[thread_index]){
        // float ilf = neighbor_ilf_hub_kernel(load_factor, initial_load_factor, rat_count, infectious_rat_count, neighbor, neighbor_start, thread_index, HUB_THREASHOLD); // INT_MAX means compute for all possible neighbors
        float ilf = neighbor_ilf_fast_kernel(load_factor, initial_load_factor, share_rat_count, share_infectious_rat_count, width, height); 
        int count = share_rat_count[in_block_id];
        result[thread_index] = mweight_kernel((float) count/load_factor, ilf);
    }
}



__global__ void find_all_sums_hub_kernel(int* hub, int nhub, float *node_weight, int *neighbor, int *neighbor_start, float *neighbor_accum_weight_result, float *sum_weight_result){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nid = hub[x];
        float sum = 0.0;
        for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) { // this eid is just index of the neighbor in the neighbor array
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}  
__global__ void find_all_sums_kernel(bool *mask, float *node_weight, int *neighbor, int *neighbor_start, float *neighbor_accum_weight_result, float *sum_weight_result, int width, int height){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id
    if (x < width && y < height && mask[nid]){
        float sum = 0.0;
        int end = min(neighbor_start[nid+1], neighbor_start[nid]+HUB_THREASHOLD+1); //+1 because HUB_THREASHOLD is out degree
        for (int eid = neighbor_start[nid]; eid < end; eid++) { // this eid is just index of the neighbor in the neighbor array
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}   




extern "C" float* compute_all_weights_cuda(state_t *s){

    graph_t *g = s->g;
    int nnode = g->nnode;
    int width = g->width;
    int height = g->height;
    int nhub = g->nhub;


    int totalBytes = sizeof(double) * nnode;
    
    double startTime = CycleTimer::currentSeconds();

    hipMemcpy(rat_count_device, s->rat_count, sizeof(int) * nnode, hipMemcpyHostToDevice);
    hipMemcpy(infectious_rat_count_device, s->infectious_rat_count, sizeof(int) * nnode, hipMemcpyHostToDevice);


    double myTime = CycleTimer::currentSeconds();

    dim3 hubBlockDim(HUB_BLOCK_SIZE);
    int hub_num_block_x = (nhub + HUB_BLOCK_SIZE - 1) / HUB_BLOCK_SIZE;
    dim3 hubGridDim(hub_num_block_x);
    compute_weight_hub_kernel<<<hubGridDim, hubBlockDim>>>(s->load_factor, initial_load_factor_device, hub_device, nhub, rat_count_device,infectious_rat_count_device, neighbor_device, neighbor_start_device, weight_result_device);
    // hipDeviceSynchronize();
    // printf("compute_weights: Overall hub: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * (CycleTimer::currentSeconds()-myTime), toBW(totalBytes, (CycleTimer::currentSeconds()-myTime)));

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    int num_block_x = (width+BLOCK_SIZE-1) / BLOCK_SIZE;
    int num_block_y = (height+BLOCK_SIZE-1) / BLOCK_SIZE;
    dim3 gridDim(num_block_x, num_block_y, 1);
    compute_weight_kernel<<<gridDim, blockDim>>>(mask_device, s->load_factor, initial_load_factor_device, rat_count_device, infectious_rat_count_device, neighbor_device, neighbor_start_device, weight_result_device, width, height);

    hipDeviceSynchronize();
    double myTimeEnd = CycleTimer::currentSeconds();
    double time_without_mem = myTimeEnd - myTime;
    // printf("compute_weights: Overall without memcpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * time_without_mem, toBW(totalBytes, time_without_mem));


    float *temp = (float *)malloc(sizeof(float) * nnode);
    hipMemcpy(temp, weight_result_device, sizeof(float) * nnode, hipMemcpyDeviceToHost);


    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    // printf("compute_weights: Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    return temp;
}

extern "C" void find_all_sums_cuda(state_t *s){
    graph_t *g = s->g;
    int nnode = g->nnode;
    int nedge = g->nedge;
    int width = g->width;
    int height = g->height;
    int nhub = g->nhub;
    double startTime = CycleTimer::currentSeconds();
    // printf("nhub ooooo, %d, outdegree %d", nhub, g->neighbor_start[g->hub[0]+1] - g->neighbor_start[g->hub[0]] - 1);

    int totalBytes = sizeof(double) * nnode;



    dim3 hubBlockDim(HUB_BLOCK_SIZE);
    int hub_num_block_x = (nhub + HUB_BLOCK_SIZE - 1) / HUB_BLOCK_SIZE;
    dim3 hubGridDim(hub_num_block_x);
    find_all_sums_hub_kernel<<<hubGridDim, hubBlockDim>>>(hub_device, nhub, weight_result_device, neighbor_device, neighbor_start_device, neighbor_accum_weight_result_device, sum_weight_result_device);

    // hipDeviceSynchronize();
    // printf("find_sums: Overall normal node: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * (CycleTimer::currentSeconds()-startTime), toBW(totalBytes, (CycleTimer::currentSeconds()-startTime)));
    


    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    int num_block_x = (width+BLOCK_SIZE-1) / BLOCK_SIZE;
    int num_block_y = (height+BLOCK_SIZE-1) / BLOCK_SIZE;
    dim3 gridDim(num_block_x, num_block_y, 1);
    find_all_sums_kernel<<<gridDim, blockDim>>>(mask_device, weight_result_device, neighbor_device, neighbor_start_device, neighbor_accum_weight_result_device, sum_weight_result_device, width, height);


    hipDeviceSynchronize();
    double myTimeEnd = CycleTimer::currentSeconds();
    double time_without_mem = myTimeEnd - startTime;
    // printf("find_sums: Overall without memcpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * time_without_mem, toBW(totalBytes, time_without_mem));

    float *temp = (float *)malloc(sizeof(float) * (nnode+nedge));
    float *temp2 = (float *)malloc(sizeof(float) * (nnode));
    hipMemcpy(temp, neighbor_accum_weight_result_device, sizeof(float) * (nnode+nedge), hipMemcpyDeviceToHost);
    hipMemcpy(temp2, sum_weight_result_device, sizeof(float) * nnode, hipMemcpyDeviceToHost);
    for (int i = 0; i < nnode+nedge; i++) {
        s->neighbor_accum_weight[i] = static_cast<double>(temp[i]);

    }
    for (int i = 0; i < nnode; i++) {
        s->sum_weight[i] = static_cast<double>(temp2[i]);
    }
    free(temp);
    free(temp2);

    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    // printf("find_sums: Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

}



void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
