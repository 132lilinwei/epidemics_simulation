#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <math.h>
#include <vector>

#include "CycleTimer.h"
#include "crun.h"
#include "rutil.h"


#define BLOCK_SIZE 16
#define HUB_BLOCK_SIZE 32

float toBW(int bytes, float sec) {
  return (float)(bytes) / (1024. * 1024. * 1024.) / sec;
}


int *hub_device;
bool *mask_device;
int *neighbor_device;
int *neighbor_start_device; 

double *initial_load_factor_device;
int *rat_count_device;
int *infectious_rat_count_device;
double* weight_result_device;

double *neighbor_accum_weight_result_device;
double *sum_weight_result_device;

extern "C" void init_cuda(state_t *s) {
    graph_t *g = s->g;
    int nnode = g->nnode;
    int nedge = g->nedge;
    int nhub = g->nhub;


    hipMalloc(&hub_device, sizeof(int) * nhub);
    hipMalloc(&mask_device, sizeof(bool) * nnode);
    hipMalloc(&neighbor_device, sizeof(int) * (nnode+nedge));
    hipMalloc(&neighbor_start_device, sizeof(int) * (nnode+1));

    hipMalloc(&initial_load_factor_device, sizeof(double) * nnode);
    hipMalloc(&rat_count_device, sizeof(int) * nnode);
    hipMalloc(&infectious_rat_count_device, sizeof(int) * nnode);
    hipMalloc(&weight_result_device, sizeof(double) * nnode);

    hipMalloc(&neighbor_accum_weight_result_device, sizeof(double) * (nnode+nedge));
    hipMalloc(&sum_weight_result_device, sizeof(double) * nnode);


    hipMemcpy(initial_load_factor_device, s->initial_load_factor, sizeof(double) * nnode, hipMemcpyHostToDevice); 
    hipMemcpy(hub_device, g->hub, sizeof(int) * nhub, hipMemcpyHostToDevice); 
    hipMemcpy(mask_device, g->mask, sizeof(bool) * nnode, hipMemcpyHostToDevice);
    hipMemcpy(neighbor_device, g->neighbor, sizeof(int) * (nnode+nedge), hipMemcpyHostToDevice);
    hipMemcpy(neighbor_start_device, g->neighbor_start,  sizeof(int) * (nnode+1), hipMemcpyHostToDevice); 

}

extern "C" void clean_cuda() {
    hipFree(hub_device);
    hipFree(mask_device);
    hipFree(neighbor_device);
    hipFree(neighbor_start_device);

    hipFree(initial_load_factor_device);
    hipFree(rat_count_device);
    hipFree(infectious_rat_count_device);
    hipFree(weight_result_device);

    hipFree(neighbor_accum_weight_result_device);
    hipFree(sum_weight_result_device);
}





__device__ __inline__ double mweight_kernel(double val, double optval) {
    double arg = 1.0 + COEFF * (val - optval);
    double lg = log(arg) * M_LOG2E;
    double denom = 1.0 + lg * lg;
    return 1.0/denom;
}

/* Compute imbalance between local and remote values */
/* Result < 0 when lcount > rcount and > 0 when lcount < rcount */
__device__ __inline__ double imbalance_kernel(int lcount, int rcount) {
    if (lcount == 0 && rcount == 0)
    return 0.0;
    double sl = sqrt((double) lcount);
    double sr = sqrt((double) rcount);
    return (sr-sl)/(sr+sl);
}

__device__ __inline__ double imbalance_density_kernel(double ldensity, double rdensity) {
    return (rdensity - ldensity) / (rdensity + ldensity);
}


__device__ __inline__ double neighbor_ilf_fast_kernel(double load_factor, double *initial_load_factor, int *rat_count, int *infectious_rat_count, int* share_rat_count, int* share_infectious_rat_count, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id
    int in_block_x = threadIdx.x;
    int in_block_y = threadIdx.y;
    int in_block_id = in_block_x + in_block_y * BLOCK_SIZE;
    double sum = 0.0;
    double ldensity, rdensity;
    int remote_x, remote_y, remote_in_block_x, remote_in_block_y, remote_nid, remote_in_block_id;

    int outdegree = 4;
    if (x == 0) {
        outdegree--;
    }
    if (y == 0) {
        outdegree--;
    }
    if (x == width -1) {
        outdegree--;
    }
    if (y == height-1) {
        outdegree--;
    }

    ldensity = (share_rat_count[in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[in_block_id] / share_rat_count[in_block_id];
    //up
    remote_x = x;
    remote_y = y+1;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x;
    remote_in_block_y = in_block_y+1;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * BLOCK_SIZE;

    if (remote_y < height) {
        if (remote_in_block_y >= BLOCK_SIZE) {
            rdensity = (rat_count[remote_nid] == 0) ? 0.0 : 1.0 * infectious_rat_count[remote_nid] / rat_count[remote_nid];
        } else {
            rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        }
        double r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }


   
    //down
    remote_x = x;
    remote_y = y-1;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x;
    remote_in_block_y = in_block_y-1;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * BLOCK_SIZE;

    if (remote_y >= 0) {
        if (remote_in_block_y < 0) {
            rdensity = (rat_count[remote_nid] == 0) ? 0.0 : 1.0 * infectious_rat_count[remote_nid] / rat_count[remote_nid];
        } else {
            rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        }
        double r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }

    //left
    remote_x = x-1;
    remote_y = y;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x-1;
    remote_in_block_y = in_block_y;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * BLOCK_SIZE;

    if (remote_x >= 0) {
        if (remote_in_block_x < 0) {
            rdensity = (rat_count[remote_nid] == 0) ? 0.0 : 1.0 * infectious_rat_count[remote_nid] / rat_count[remote_nid];
        } else {
            rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        }
        double r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }

    //right
    remote_x = x+1;
    remote_y = y;
    remote_nid = remote_x + remote_y * width;
    remote_in_block_x = in_block_x+1;
    remote_in_block_y = in_block_y;
    remote_in_block_id = remote_in_block_x + remote_in_block_y * BLOCK_SIZE;

    if (remote_x <width) {
        if (remote_in_block_x >= BLOCK_SIZE) {
            rdensity = (rat_count[remote_nid] == 0) ? 0.0 : 1.0 * infectious_rat_count[remote_nid] / rat_count[remote_nid];
        } else {
            rdensity = (share_rat_count[remote_in_block_id] == 0) ? 0.0 : 1.0 * share_infectious_rat_count[remote_in_block_id] / share_rat_count[remote_in_block_id];
        }
        double r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 :imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }

    // change to a new ilf, where each node has different initial base ilf
    double ilf = BASE_ILF * (initial_load_factor[nid] / load_factor) + ILF_VARIABILITY * (sum / outdegree);
    return ilf;
}


__device__ __inline__ double neighbor_ilf_hub_kernel(double load_factor, double *initial_load_factor, int *rat_count, int *infectious_rat_count, int *neighbor, int *neighbor_start, int nid, int max_outdegree) {
    int outdegree = neighbor_start[nid+1] - neighbor_start[nid] - 1;
    outdegree = min(outdegree, max_outdegree);
    int *start = &neighbor[neighbor_start[nid]+1];
    int i;
    double sum = 0.0;
    for (i = 0; i < outdegree; i++) {
       
        double ldensity = (rat_count[nid] == 0) ? 0.0 : 1.0 * infectious_rat_count[nid] / rat_count[nid];
        double rdensity = (rat_count[start[i]] == 0) ? 0.0 : 1.0 * infectious_rat_count[start[i]] / rat_count[start[i]];
        double r = (ldensity == 0.0 && rdensity == 0.0) ? 0.0 : imbalance_density_kernel(ldensity, rdensity);
        sum += r;
    }
    // change to a new ilf, where each node has different initial base ilf
    double ilf = BASE_ILF * (initial_load_factor[nid] / load_factor) + ILF_VARIABILITY * (sum / outdegree);
    return ilf;
}



__global__ void
compute_weight_hub_kernel(double load_factor, double *initial_load_factor, int* hub, int nhub, int *rat_count, int *infectious_rat_count, int *neighbor, int *neighbor_start, double* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nodeid = hub[x];
        double ilf = neighbor_ilf_hub_kernel(load_factor, initial_load_factor, rat_count, infectious_rat_count, neighbor, neighbor_start, nodeid, INT_MAX); // INT_MAX means compute for all possible neighbors
        int count = rat_count[nodeid];
        result[nodeid] = mweight_kernel((double) count/load_factor, ilf);
    }
}



__global__ void
compute_weight_kernel(bool *mask, double load_factor, double *initial_load_factor, int *rat_count, int *infectious_rat_count, int *neighbor, int *neighbor_start, double* result, int width, int height) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thread_index = y * width + x; // thread_index is node id

    int in_block_x = threadIdx.x;
    int in_block_y = threadIdx.y;
    int in_block_id = in_block_x + in_block_y * BLOCK_SIZE;


    __shared__ int share_rat_count[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ int share_infectious_rat_count[BLOCK_SIZE * BLOCK_SIZE];

    share_rat_count[in_block_id] = rat_count[thread_index];
    share_infectious_rat_count[in_block_id] = infectious_rat_count[thread_index];



    if (x < width && y < height && mask[thread_index]){
        // double ilf = neighbor_ilf_hub_kernel(load_factor, initial_load_factor, rat_count, infectious_rat_count, neighbor, neighbor_start, thread_index, HUB_THREASHOLD); // INT_MAX means compute for all possible neighbors
        double ilf = neighbor_ilf_fast_kernel(load_factor, initial_load_factor, rat_count, infectious_rat_count, share_rat_count, share_infectious_rat_count, width, height); 
        int count = share_rat_count[in_block_id];
        result[thread_index] = mweight_kernel((double) count/load_factor, ilf);
    }
}



__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nid = hub[x];
        double sum = 0.0;
        for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) { // this eid is just index of the neighbor in the neighbor array
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}  
__global__ void find_all_sums_kernel(bool *mask, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result, int width, int height){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id
    if (x < width && y < height && mask[nid]){
        double sum = 0.0;
        int end = min(neighbor_start[nid+1], neighbor_start[nid]+HUB_THREASHOLD+1); //+1 because HUB_THREASHOLD is out degree
        for (int eid = neighbor_start[nid]; eid < end; eid++) { // this eid is just index of the neighbor in the neighbor array
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}   




extern "C" void compute_all_weights_cuda(state_t *s){

    graph_t *g = s->g;
    int nnode = g->nnode;
    int width = g->width;
    int height = g->height;
    int nhub = g->nhub;


    int totalBytes = sizeof(double) * nnode;
    
    double startTime = CycleTimer::currentSeconds();

    hipMemcpy(rat_count_device, s->rat_count, sizeof(int) * nnode, hipMemcpyHostToDevice);
    hipMemcpy(infectious_rat_count_device, s->infectious_rat_count, sizeof(int) * nnode, hipMemcpyHostToDevice);


    double myTime = CycleTimer::currentSeconds();

    dim3 hubBlockDim(HUB_BLOCK_SIZE);
    int hub_num_block_x = (nhub + HUB_BLOCK_SIZE - 1) / HUB_BLOCK_SIZE;
    dim3 hubGridDim(hub_num_block_x);
    compute_weight_hub_kernel<<<hubGridDim, hubBlockDim>>>(s->load_factor, initial_load_factor_device, hub_device, nhub, rat_count_device,infectious_rat_count_device, neighbor_device, neighbor_start_device, weight_result_device);
    // hipDeviceSynchronize();
    // printf("compute_weights: Overall hub: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * (CycleTimer::currentSeconds()-myTime), toBW(totalBytes, (CycleTimer::currentSeconds()-myTime)));

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    int num_block_x = (width+BLOCK_SIZE-1) / BLOCK_SIZE;
    int num_block_y = (height+BLOCK_SIZE-1) / BLOCK_SIZE;
    dim3 gridDim(num_block_x, num_block_y, 1);
    compute_weight_kernel<<<gridDim, blockDim>>>(mask_device, s->load_factor, initial_load_factor_device, rat_count_device, infectious_rat_count_device, neighbor_device, neighbor_start_device, weight_result_device, width, height);

    hipDeviceSynchronize();
    double myTimeEnd = CycleTimer::currentSeconds();
    double time_without_mem = myTimeEnd - myTime;
    // printf("compute_weights: Overall without memcpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * time_without_mem, toBW(totalBytes, time_without_mem));



    hipMemcpy(s->node_weight, weight_result_device, sizeof(double) * nnode, hipMemcpyDeviceToHost);


    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    // printf("compute_weights: Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
}

extern "C" void find_all_sums_cuda(state_t *s){
    graph_t *g = s->g;
    int nnode = g->nnode;
    int nedge = g->nedge;
    int width = g->width;
    int height = g->height;
    int nhub = g->nhub;
    double startTime = CycleTimer::currentSeconds();
    // printf("nhub ooooo, %d, outdegree %d", nhub, g->neighbor_start[g->hub[0]+1] - g->neighbor_start[g->hub[0]] - 1);

    int totalBytes = sizeof(double) * nnode;



    dim3 hubBlockDim(HUB_BLOCK_SIZE);
    int hub_num_block_x = (nhub + HUB_BLOCK_SIZE - 1) / HUB_BLOCK_SIZE;
    dim3 hubGridDim(hub_num_block_x);
    find_all_sums_hub_kernel<<<hubGridDim, hubBlockDim>>>(hub_device, nhub, weight_result_device, neighbor_device, neighbor_start_device, neighbor_accum_weight_result_device, sum_weight_result_device);

    // hipDeviceSynchronize();
    // printf("find_sums: Overall normal node: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * (CycleTimer::currentSeconds()-startTime), toBW(totalBytes, (CycleTimer::currentSeconds()-startTime)));
    


    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    int num_block_x = (width+BLOCK_SIZE-1) / BLOCK_SIZE;
    int num_block_y = (height+BLOCK_SIZE-1) / BLOCK_SIZE;
    dim3 gridDim(num_block_x, num_block_y, 1);
    find_all_sums_kernel<<<gridDim, blockDim>>>(mask_device, weight_result_device, neighbor_device, neighbor_start_device, neighbor_accum_weight_result_device, sum_weight_result_device, width, height);


    hipDeviceSynchronize();
    double myTimeEnd = CycleTimer::currentSeconds();
    double time_without_mem = myTimeEnd - startTime;
    // printf("find_sums: Overall without memcpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * time_without_mem, toBW(totalBytes, time_without_mem));

    hipMemcpy(s->neighbor_accum_weight, neighbor_accum_weight_result_device, sizeof(double) * (nnode+nedge), hipMemcpyDeviceToHost);
    hipMemcpy(s->sum_weight, sum_weight_result_device, sizeof(double) * nnode, hipMemcpyDeviceToHost);

    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    // printf("find_sums: Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

}



void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
